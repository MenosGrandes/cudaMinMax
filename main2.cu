#include "hip/hip_runtime.h"
#ifndef __HIPCC__
    #define __HIPCC__
#endif
#include <math.h>
#include <stdio.h>
#include <random>
#include <iomanip>
#include <iostream>
#include <algorithm>
#include <random>
#include "random.hpp"
#include "helpers.cu"
#include "kernel.cu"
int  minmaxCuda(double* max, double* min, const double* a)
{
    double* dev_a = 0;
    double* dev_max = 0;
    double* dev_min = 0;
    dim3 dimBlock(BLOCKSIZE);
    dim3 dimGrid(SIZE);

    if (
        (hipSetDevice(0) != hipSuccess) ||
        (hipMalloc((void**)&dev_max, SIZE * sizeof(double)) != hipSuccess) ||
        (hipMalloc((void**)&dev_min, SIZE * sizeof(double)) != hipSuccess) ||
        (hipMalloc((void**)&dev_a, SIZE * SIZE  * sizeof(double)) != hipSuccess) ||
        (hipMemcpy(dev_a, a, SIZE * SIZE * sizeof(double), hipMemcpyHostToDevice) != hipSuccess)
    )
    {
        fprintf(stderr, "hipSetDevice failed/ hipMalloc failed/hipMemcpy failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    TIMERSTART(SeqMinMaxKernel);
    seq_minmaxKernel <<< dimGrid, dimBlock>>>(dev_max, dev_min, dev_a);
    hipDeviceSynchronize();
    seq_finalminmaxKernel <<< 1, dimBlock>>>(dev_max, dev_min);
    hipDeviceSynchronize();
    TIMERSTOP(SeqMinMaxKernel);
    CUERR;

    if (hipDeviceSynchronize() != hipSuccess)
    {
        fprintf(stderr, "hipDeviceSynchronize returned error code  after launching minmaxKernel!\n");
        goto Error;
    }

    if ((hipMemcpy(max, dev_max, SIZE * sizeof(double), hipMemcpyDeviceToHost) != hipSuccess) ||
        (hipMemcpy(min, dev_min, SIZE * sizeof(double), hipMemcpyDeviceToHost) != hipSuccess)
       )
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    return 0;
Error:
    hipFree(dev_max);
    hipFree(dev_min);
    hipFree(dev_a);
    return -1;
}

int main()
{
    Random < double, SIZE, -1, 1 > r;
    double* a = r.generateRandomArray();
    double* max = new double[SIZE];
    double* min = new double[SIZE];

    if (minmaxCuda(max, min, a))
    {
        fprintf(stderr, "minmaxCuda failed!");
        return 1;
    }

#ifdef DEBUG
    std::pair<float, float> minMax;

    for (int i = 0; i < SIZE * SIZE; i++)
    {
        if (a[i] < minMax.first)
        {
            minMax.first = a[i];
        }
        else if (a[i] > minMax.second)
        {
            minMax.second = a[i];
        }

        //std::cout << a[i] << " ";
    }

    std::cout << std::endl;
    std::cout << "CPU MIN: " << minMax.first << " MAX: " << minMax.second << std::endl;
    std::cout << "GPU MIN: " << min[0] << " MAX: " << max[0] << std::endl;
#endif

    if (hipDeviceReset() != hipSuccess)
    {
        fprintf(stderr, "hipDeviceReset failed!");
        delete []a;
        delete []min;
        delete []max;
        return 1;
    }

    delete []a;
    delete []min;
    delete []max;
    return 0;
}


